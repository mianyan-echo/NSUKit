#include "hip/hip_runtime.h"
////////////////////////////////////////////////////////////////////////////////
// Copyright (c) 2024. Naishu
// NSUKit is licensed under Mulan PSL v2.
// You can use this software according to the terms and conditions of the Mulan PSL v2.
// You may obtain a copy of Mulan PSL v2 at:
//          http://license.coscl.org.cn/MulanPSL2
// THIS SOFTWARE IS PROVIDED ON AN "AS IS" BASIS, WITHOUT WARRANTIES OF ANY KIND,
// EITHER EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO NON-INFRINGEMENT,
// MERCHANTABILITY OR FIT FOR A PARTICULAR PURPOSE.
// See the Mulan PSL v2 for more details.
////////////////////////////////////////////////////////////////////////////////

//
// Created by jilianyi<jilianyi@naishu.tech> on 2024/5/8.
//


#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime.h>
#include "NSUKit.h"


/**
 * algo example
 * @param ptr p_device_mem
 */
__global__ void cu_example_kernel(uint32_t *ptr) {
    unsigned int i = threadIdx.x;
    unsigned int step = blockDim.x;
    unsigned int b = blockIdx.x;

    unsigned int idx = (b*step+i)&0xFFFFFFFF;
    ptr[idx] = idx;
}


/**
 * Convenience function for checking CUDA runtime API results
 * can be wrapped around any runtime API call. No-op in release builds.
 * @param result
 * @return
 */
inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
        throw std::runtime_error("CUDA Runtime Error");
    }
    return result;
}


int main(int argc, char **argv) {
    nsuMemory_p p_dma_obj;
    void *p_host_mem, *p_gpu_mem, *p_stream_mem;
    nsuSize_t mem_size = 102400;   // 102400 Bytes
    nsukit::NSUSoc<nsukit::TCPCmdUItf, nsukit::PCIECmdUItf, nsukit::PCIEStreamUItf> kit{};
    nsuInitParam_t param;
    param.stream_board = 0;

    auto res = kit.link_stream(&param);
    if (res != nsukitStatus_t::NSUKIT_STATUS_SUCCESS) {
        std::cout << "建立DS连接：" << nsukit::status2_string(res) << std::endl;
        return 1;
    }

    checkCuda(hipHostMalloc((void **) &p_host_mem, mem_size));                         // alloc host memory with cuda api

    p_dma_obj = kit.alloc_buffer(mem_size, p_host_mem);                                 // use cuda memory as stream memory
    p_stream_mem = kit.get_buffer(p_dma_obj, mem_size);                                 // get stream memory ptr
    checkCuda(hipHostGetDevicePointer((void **) &p_gpu_mem, (void *) p_host_mem, 0));  // get gpu memory ptr

    std::cout << "host mem_ptr: "      << p_host_mem
              << " \ngpu mem_ptr: "    << p_gpu_mem
              << " \nstream mem_ptr: " << p_stream_mem << std::endl;

    // stream upload
    auto s = kit.open_recv(0, p_dma_obj, mem_size, 0);
    if (s != nsukitStatus_t::NSUKIT_STATUS_SUCCESS) {
        std::cout << "Establish CS and CR connections: "  << std::endl;
        return 1;
    }
    s = nsukitStatus_t::NSUKIT_STATUS_STREAM_RUNNING;
    while (s==nsukitStatus_t::NSUKIT_STATUS_STREAM_RUNNING) {
        s = kit.wait_stream(p_dma_obj, 1.);
    }

    // run cuda kernel without copy to gpu memory
    cu_example_kernel<<<1, 4>>>((uint32_t *) p_gpu_mem);
    hipDeviceSynchronize();

    return 0;
}
